#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// 寻找给定无符号整数 n 的最高有效位（Most Significant Bit, MSB）的下一个最高位
// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// 对于边界矩形重叠的每个瓦片，具有一个键/值对。
		// 键是 | 瓦片ID | 深度 |，
		// 值是高斯的ID，按照这个键对值进行排序，将得到一个高斯ID列表，
		// 这样它们首先按瓦片排序，然后按深度排序
		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// 识别每个瓦片（tile）在排序后的高斯ID列表中的范围
// 目的是确定哪些高斯ID属于哪个瓦片，并记录每个瓦片的开始和结束位置
// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// 在给定的内存块中初始化 GeometryState 结构
// chunk（一个指向内存块的指针引用），P（元素的数量）
// 使用 obtain 函数为 GeometryState 的不同成员分配空间，并返回一个初始化的 GeometryState 实例
CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

// 初始化 BinningState 实例，分配所需的内存，并执行排序操作
CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	// 在 GPU 上进行基数排序, 将 point_list_keys_unsorted 作为键，point_list_unsorted 作为值进行排序，排序结果存储在 point_list_keys 和 point_list 中
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);  // 垂直方向的焦距 focal_y
	const float focal_x = width / (2.0f * tan_fovx);   // 水平方向的焦距 focal_x

	size_t chunk_size = required<GeometryState>(P);  // 计算存储所有3D gaussian的各个参数所需要的空间大小
	char* chunkptr = geometryBuffer(chunk_size);  // 给所有3D gaussian的各个参数分配存储空间, 并返回存储空间的指针
	// 在给定的内存块中初始化 GeometryState 结构体, 为不同成员分配空间，并返回一个初始化的实例
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;  // 指向radii数据的指针
	}
	
	// 定义了一个三维网格（dim3 是 CUDA 中定义三维网格维度的数据类型），确定了在水平和垂直方向上需要多少个块来覆盖整个渲染区域
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	// 确定了每个块在 X（水平）和 Y（垂直）方向上的线程数
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	// 计算存储所有2D pixel的各个参数所需要的空间大小
	size_t img_chunk_size = required<ImageState>(width * height);
	// 给所有2D pixel的各个参数分配存储空间, 并返回存储空间的指针
	char* img_chunkptr = imageBuffer(img_chunk_size);
	// 在给定的内存块中初始化 ImageState 结构体, 为不同成员分配空间，并返回一个初始化的实例
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,  // 3D gaussian的个数, 球谐函数的次数, 球谐系数的个数 (球谐系数用于表示颜色)
		means3D,  // 每个3D gaussian的XYZ均值
		(glm::vec3*)scales,  // 每个3D gaussian的XYZ尺度
		scale_modifier,   // 尺度缩放系数, 1.0
		(glm::vec4*)rotations,  // 每个3D gaussian的旋转四元组
		opacities,    // 每个3D gaussian的不透明度
		shs,   // 每个3D gaussian的球谐系数, 用于表示颜色
		geomState.clamped,  // 存储每个3D gaussian的R、G、B是否小于0
		cov3D_precomp,   // 提前计算好的每个3D gaussian的协方差矩阵, []
		colors_precomp,  // 提前计算好的每个3D gaussian的颜色, []
		viewmatrix,   // 相机外参矩阵, world to camera
		projmatrix,   // 投影矩阵, world to image
		(glm::vec3*)cam_pos,  // 所有相机的中心点XYZ坐标
		width, height,        // 图像的宽和高
		focal_x, focal_y,     // 水平、垂直方向的焦距
		tan_fovx, tan_fovy,   // 水平、垂直视场角一半的正切值
		radii,  // 存储每个2D gaussian在图像上的半径
		geomState.means2D,   // 存储每个2D gaussian的均值
		geomState.depths,    // 存储每个2D gaussian的深度
		geomState.cov3D,     // 存储每个3D gaussian的协方差矩阵
		geomState.rgb,       // 存储每个2D pixel的颜色
		geomState.conic_opacity,  // 存储每个2D gaussian的协方差矩阵的逆矩阵以及它的不透明度
		tile_grid,  // 在水平和垂直方向上需要多少个块来覆盖整个渲染区域
		geomState.tiles_touched,  // 存储每个2D gaussian覆盖了多少个tile
		prefiltered    // 是否预先过滤掉了中心点(均值XYZ)不在视锥（frustum）内的3D gaussian
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;  // 存储所有的2D gaussian总共覆盖了多少个tile
	// 将 geomState.point_offsets 数组中最后一个元素的值复制到主机内存中的变量 num_rendered
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// 将每个3D gaussian的对应的tile index和深度存到point_list_keys_unsorted中
    // 将每个3D gaussian的对应的index（第几个3D gaussian）存到point_list_unsorted中
	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
	CHECK_CUDA(, debug)

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// 对一个键值对列表进行排序。这里的键值对由 binningState.point_list_keys_unsorted 和 binningState.point_list_unsorted 组成
    // 排序后的结果存储在 binningState.point_list_keys 和 binningState.point_list 中
    // binningState.list_sorting_space 和 binningState.sorting_size 指定了排序操作所需的临时存储空间和其大小
    // num_rendered 是要排序的元素总数。0, 32 + bit 指定了排序的最低位和最高位，这里用于确保排序考虑到了足够的位数，以便正确处理所有的键值对
	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,
		binningState.sorting_size,
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug)

	// 将 imgState.ranges 数组中的所有元素设置为 0
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// 识别每个瓦片（tile）在排序后的高斯ID列表中的范围
    // 目的是确定哪些高斯ID属于哪个瓦片，并记录每个瓦片的开始和结束位置
	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid,   // 在水平和垂直方向上需要多少个块来覆盖整个渲染区域
		block,  // 每个块在 X（水平）和 Y（垂直）方向上的线程数
		imgState.ranges,  // 每个瓦片（tile）在排序后的高斯ID列表中的范围
		binningState.point_list,  // 排序后的3D gaussian的id列表
		width, height,  // 图像的宽和高
		geomState.means2D,  // 每个2D gaussian在图像上的中心点位置
		feature_ptr,   // 每个3D gaussian对应的RGB颜色
		geomState.conic_opacity,  // 每个2D gaussian的协方差矩阵的逆矩阵以及它的不透明度
		imgState.accum_alpha,  // 渲染过程后每个像素的最终透明度或透射率值
		imgState.n_contrib,  // 每个pixel的最后一个贡献的2D gaussian是谁
		background,  // 背景颜色
		out_color), debug)  // 输出图像

	return num_rendered;
}

// 产生对应于前向渲染过程所需的优化梯度
// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);
	
	// 根据每像素损失梯度计算损失梯度，关于2D均值位置、圆锥矩阵、
	// 高斯的不透明度和RGB。如果我们获得了预计算的颜色而不是球谐系数，就使用它们。
	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// 处理预处理的剩余部分
	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}